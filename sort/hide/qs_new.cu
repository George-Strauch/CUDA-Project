#include "hip/hip_runtime.h"
#include <iostream>

/*
written by George Strauch on 4/21/2020

c++ program to sort an array with quicksort algorithm

Execution follows the syntax:
$ ./exec {int num of elements}

Example run:
$ nvcc qs.cu -arch='sm_35' -rdc=true -lcudadevrt -o gpu_q
$ time ./gpu_qs 10
$ time ./gpu_qs 999
*/



__host__  // used for debug
void print_array (int *array, int n, int tag_index)
{
  for (size_t i = 0; i < n; i++) {
    if (i == tag_index+1) {
      std::cout << " > ";
    }
    std::cout << array[i] << ' ';
  }
  std::cout << '\n';
}



__host__
int* allocate_shared_array(int n_elements)
{
  int *a;
  hipMallocManaged(&a, n_elements*sizeof(int));
  return a;
}



__host__  // makes and returns unsorted array with random elements
int* make_unsorted_array(int n_elements)
{
  int *a = allocate_shared_array(n_elements);
  for (size_t j = 0; j < n_elements; j++) {
    a[j] =  rand()%(2*n_elements);
  }
  return a;
}



__device__  // helper function to overwrite section of array
void overwrite(int* &new_array, int* original, int n)
{
  for (size_t i = 0; i < n; i++) {
    original[i] = new_array[i];
  }
  hipFree(new_array);
}



// __global__
// void sort(int* array, int n)
// {
//   // dont do anything if array size is 0 or 1
//   if (n < 2) { return; }
//   int *tmparray;
//   hipMalloc(&tmparray, n*sizeof(int));
//
//   int piv = array[n-1];
//   int lower_or_equal = 0;   // num of elements lower or equal to piv
//   int higher = 0;           // num of elements higher to piv
//
//   // if element lower or equal to piv, append to bottom of new array
//   // else, apend to top
//   // then overwite array with new_array
//   for (size_t i = 0; i < n; i++) {
//     if(array[i] <= piv){
//       tmparray[lower_or_equal] = array[i];
//       lower_or_equal++;
//     }
//     else {
//       tmparray[n-higher-1] = array[i];
//       higher++;
//     }
//   }
//   overwrite(tmparray, array, n);
//
//   // if no elements are higher than piv, piv remains at top, so sort bottom n-1
//   if (higher == 0) {
//     sort<<<1,1>>>(array, lower_or_equal-1);
//   }
//   else {
//     sort<<<1,1>>>(array, lower_or_equal);
//     sort<<<1,1>>>(&array[lower_or_equal], higher);
//   }
//   hipDeviceSynchronize();
// }












__global__
void sort(int* array, int l, int h)
{
  // dont do anything if array size is 0 or 1
  if (l+h < 2) { return; }

  int n;

  if (threadIdx.x == 0){
    array = array;
    n = l;
  }
  else {
    array = &array[l];
    n = h-l;
  }

  int *tmparray;
  hipMalloc(&tmparray, n*sizeof(int));

  int piv = array[n-1];
  int lower_or_equal = 0;   // num of elements lower or equal to piv
  int higher = 0;           // num of elements higher to piv

  // if element lower or equal to piv, append to bottom of new array
  // else, apend to top
  // then overwite array with new_array
  for (size_t i = 0; i < n; i++) {
    if(array[i] <= piv){
      tmparray[lower_or_equal] = array[i];
      lower_or_equal++;
    }
    else {
      tmparray[n-higher-1] = array[i];
      higher++;
    }
  }
  overwrite(tmparray, array, n);

  // if no elements are higher than piv, piv remains at top, so sort bottom n-1
  if (higher == 0) {
    sort<<<1,1>>>(array, lower_or_equal, higher);
  }
  else {
    sort<<<1,2>>>(array, lower_or_equal, higher);
  }

  hipDeviceSynchronize();
}









__host__  // retruuns false if any element larger than i+1 element
int verify_in_order(int* array, int n)
{
  for (size_t i = 0; i < n-1; i++) {
    if (array[i+1] < array[i]) {
      std::cout << "\nindex: " << i << '\n';
      return i;
    }
  }
  return -1;
}



int main(int argc, char const *argv[])
{

  int N = atoi(argv[1]);
  std::cout << "N = " << N << '\n';

  int* a = make_unsorted_array(N);

  // sort(array, lower_or_equal, higher, n-1);
  sort<<<1,1>>>(a, N, 0);
  hipDeviceSynchronize();

  int order = verify_in_order(a, N);

  if (order == -1) {
    std::cout << "array is in order" << '\n';
  }
  else {
    std::cout << "not in order"  << '\n';
    print_array(a, N, order);
  }

  hipFree(a);
  return 0;
}


















//
