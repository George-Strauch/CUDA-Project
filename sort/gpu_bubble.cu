#include <iostream>
#include <hip/hip_runtime.h>

// includes CUDA Runtime

#include <hip/hip_runtime_api.h>

/*
written by George Strauch on 4/21/2020

c++ program to sort an array with quicksort algorithm

Execution follows the syntax:
$ ./exec {int num of elements}

Example run:
$ nvcc gpu_bubble.cu -arch='sm_35' -rdc=true -lcudadevrt -o gpu_q
or $ nvcc -lineinfo -arch=sm_35 gpu_bubble.cu -o g
$ time ./gpu_qs 10
$ time ./gpu_qs 999
*/



__host__  // used for debug
void print_array (int *array, int n, int tag_index)
{
  for (size_t i = 0; i < n; i++) {
    if (i == tag_index+1) {
      std::cout << " > ";
    }
    std::cout << array[i] << ' ';
  }
  std::cout << '\n';
}



__host__
int* allocate_shared_array(int n_elements)
{
  int *a;
  hipMallocManaged(&a, n_elements*sizeof(int));
  return a;
}



__host__  // makes and returns unsorted array with random elements
int* make_unsorted_array(int n_elements)
{
  int *a = allocate_shared_array(n_elements);
  for (size_t j = 0; j < n_elements; j++) {
    a[j] =  rand()%(2*n_elements);
  }
  return a;
}



__host__
bool go_again(int* array, int n)
{
  for (size_t i = 0; i < n-1; i++) {
    if(array[i] > array[i+1])
    {
      return true;
    }
  }
  return false;
}



__global__
void sort(int* array, int n, int offset)
{
  int id = 2*threadIdx.x + offset;
  if (id >= n-1) {
    return;
  }

  int tmp;
  if (array[id] > array[id+1]) {
    tmp = array[id+1];
    array[id+1] = array[id];
    array[id] = tmp;
  }
  __syncthreads();

}



__host__  // returns element index if any element larger than i+1 element, else -1
int verify_in_order(int* array, int n)
{
  for (size_t i = 0; i < n-1; i++) {
    if (array[i+1] < array[i]) {
      return i;
    }
  }
  return -1;
}


__host__
void entry_point(int* array, int n)
{
  while (go_again(array, n)) {
    sort<<<1, n/2>>>(array, n, 0);
    sort<<<1, n/2>>>(array, n, 1);
    hipDeviceSynchronize();
  }
}



int main(int argc, char const *argv[])
{

  int N = atoi(argv[1]);
  std::cout << "N = " << N << '\n';

  int* a = make_unsorted_array(N);

  hipProfilerStart();

  // while (go_again(a, N)) {
  //   sort<<<1, N/2>>>(a, N, 0);
  //   sort<<<1, N/2>>>(a, N, 1);
  //   cudaDeviceSynchronize();
  // }
  entry_point(a, N);
  hipProfilerStop();

  int order = verify_in_order(a, N);

  if (order == -1) {
    std::cout << "array is in order" << '\n';
  }
  else {
    std::cout << "not in order"  << '\n';
    print_array(a, N, order);
  }

  hipFree(a);
  return 0;
}


















//
