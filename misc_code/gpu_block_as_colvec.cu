#include <iostream>
// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// maybe you need also helpers

/*
written by George Strauch on 4/19/2020

c++ program for matrix multiply using 1d arrays on the GPU
the GPU makes use of parallelism to make processes like this much faster

This implementation only uses square matrices as they are much
easier to debug, calculate and work with, however all functions can work with
non-square matrices too.

this implementation uses a block for every column and every element is computed
by a different thread

This program uses shared memory between the host CPU and the GPU.
using dedicated device memory can make the program run faster however its
make it much more difficult to work with certain datatypes such as the
struct to represent a matrix.

Execution follows the syntax:
$ ./exec {int matrix_size} {int print_option}

where the print option can be:
1: Prints the whole of each matrix for debugging
and best used with smaller matrices <= 10.
2: Shows only the first and last element of the result.
other or no option: does not print anything.

Example run:
$ nvcc gpu_mm.cu -o gpu  //-lcuda
$ time ./gpu 10 1
$ time ./gpu 1000 2
$ sudo nvprof --unified-memory-profiling off ./gpu 500 2
*/



typedef long long int lli;

// struct to make working with matrices much easier
struct Matrix
{
  lli *values;
  int rows;
  int cols;
};



// fills a matrix with values
__host__
void fillMat(Matrix m)
{
  for (size_t j = 0; j < m.rows*m.cols; j++) {
    m.values[j] = j% m.cols;
  }
}



// get a Matrix object with shared memory that can be accessed by the device
__host__
Matrix get_shared(int rows, int cols)
{
  Matrix *m;
  hipMallocManaged(&m, sizeof(Matrix));
  hipMallocManaged(&m->values, rows*cols*sizeof(lli));
  m->cols = cols;
  m->rows = rows;
  return *m;
}



// calculate a single element of the matrix result of m1*m2
// res_x = res cols = m2 cols  max
// res_y = res rows = m1 rows  max
// common = m1_cols and m2_rows
__global__
void matmul(Matrix m1, Matrix m2, Matrix res)
{
  int id = threadIdx.x*blockDim.x + blockIdx.x;
  res.values[id] = 0;
  for (size_t i = 0; i < m1.cols; i++) {
    res.values[id] += m1.values[threadIdx.x*blockDim.x+i] * m2.values[(i*m2.cols)+blockIdx.x];
  }
}



// host side function to display matrix
__host__
void displayMatrix(Matrix mat)
{
  for (size_t i = 0; i < mat.rows; i++) {
    for (size_t j = 0; j < mat.cols; j++) {
      std::cout << mat.values[i*mat.cols + j] << ' ';
    }
    std::cout << '\n';
  }
  std::cout << '\n';
}



// frees memory
__host__
void free_matrix(Matrix mat)
{
  hipFree(mat.values);
}



// returns a copy of a matrix
__host__
Matrix copyMatrix(Matrix m)
{
  Matrix nm = get_shared(m.rows, m.cols);
  for (size_t i = 0; i < m.cols*m.rows; i++) {
    nm.values[i] = m.values[i];
  }
  return nm;
}



// host side function to transpose
__host__
void transpose(Matrix &mat)
{
  Matrix new_mat = get_shared(mat.cols, mat.rows);
  for (size_t a = 0; a < mat.rows; a++) {
    for (size_t b = 0; b < mat.cols; b++) {
      new_mat.values[b*mat.cols + a] = mat.values[a*mat.cols + b];
    }
  }

  free_matrix(mat);
  mat = new_mat;
}



int main(int argc, char const *argv[])
{
  // gets the matrix size from user, see header
  int N = atoi(argv[1]);
  std::cout << "N: " << N << '\n';
  // cudaProfilerStart();
  Matrix t1 = get_shared(N, N);
  fillMat(t1);
  Matrix t2 = copyMatrix(t1);
  transpose(t2);

  Matrix res = get_shared(t1.rows, t2.cols);

  // options for building the block grid. Subject to Change.
  //------------------------------------------
  int threads_in_block = res.rows;
  int blocks = res.cols;
  //------------------------------------------


  // performs matrix multiply on the GPU, each thread will handle one element
  // then copys the result to host memory
  std::cout << "\nstart" << '\n';
  matmul<<<blocks, threads_in_block>>>(t1, t2, res);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  std::cout << "done\n" << '\n';

  // display array
  // display options listed in header
  if (argc > 2) {
    if (atoi(argv[2]) == 1) {
      std::cout << "matrix 1: " << '\n';
      displayMatrix(t1);

      std::cout << "matrix 2: " << '\n';
      displayMatrix(t2);

      std::cout << "result: " << '\n';
      displayMatrix(res);
    }
    else if (atoi(argv[2]) == 2) {
      std::cout << "first: " << res.values[0] << '\n';
      std::cout << "last: " << res.values[N*N-1] << '\n';
      std::cout << '\n';
    }
  }

  free_matrix(t1);
  free_matrix(t2);
  free_matrix(res);

  // cudaProfilerStop();

  return 0;
}














//
